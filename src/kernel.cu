#include "hip/hip_runtime.h"
#include "kernel.cuh"

#include <cuda/std/array>
#if defined(USE_SEMAPHORE)
#include <cuda/semaphore>
#endif
#include <hiprand/hiprand_kernel.h>

#include "Circle.cuh"

__constant__ Point GLOBAL_POINTS[GLOBAL_ARRAY_SIZE];
__constant__ size_t GLOBAL_POINTS_SIZE;
__constant__ float ERROR;

__device__ static cuda::std::array<Point,3> getRandomNumber();
__device__ static void count(Circle& circle);
#if defined(USE_SEMAPHORE)
__device__  cuda::binary_semaphore<cuda::thread_scope_device> binarySemaphore{1};
__device__ static void max(Circle* bestCircle, const Circle& circle);
#endif

__global__ void ransac_kernel(Circle* bestCircle) {
    const cuda::std::array<Point,3>  randomPoints = getRandomNumber();
    Circle circle = Circle::CircleFromThreePoints(randomPoints);
    count(circle);
#if defined(USE_SEMAPHORE)
    max(bestCircle, circle);
#else
    unsigned id = threadIdx.x + blockIdx.x * blockDim.x;
    bestCircle[id] = circle;
#endif
}

__device__ static cuda::std::array<Point,3> getRandomNumber() {
    hiprandState state;
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(clock64(), id, 0, &state);
    int index1 = -1, index2 = -1, index3 = -1;

    index1 = hiprand(&state) % GLOBAL_POINTS_SIZE;

    do{
        index2 = hiprand(&state) % GLOBAL_POINTS_SIZE;
    } while (index1 == index2);

    do {
        index3 = hiprand(&state) % GLOBAL_POINTS_SIZE;
    } while (index3 == index1 || index3 == index2);

    return {
            GLOBAL_POINTS[index1],
            GLOBAL_POINTS[index2],
            GLOBAL_POINTS[index3]
    };
}

__device__ static void count(Circle& circle){
    int db = 0;
    for(size_t i = 0; i < GLOBAL_POINTS_SIZE; ++i){
        if (circle.is_point_supported(GLOBAL_POINTS[i], ERROR)) ++db;
    }
    circle.setSupportedPoints(db);
}

#if defined(USE_SEMAPHORE)
__device__ static void max(Circle* bestCircle, const Circle& circle){
    binarySemaphore.acquire();

    if (bestCircle->getSupportedPoints() < circle.getSupportedPoints()){
        *bestCircle = circle;
    }

    binarySemaphore.release();
}
#else
__global__ void max_search(Circle* circles){
    extern __shared__ Circle data[];
    unsigned global_id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned local_id = threadIdx.x;

    data[local_id] = circles[global_id];
    __syncthreads();

    for (unsigned stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (local_id < stride) {
            unsigned left = local_id;
            unsigned right = left + stride;

            if (data[right].getSupportedPoints() > data[left].getSupportedPoints()){
                data[left] = data[right];
            }
        }
        __syncthreads();
    }

    if (local_id == 0) {
        circles[blockIdx.x] = data[0];
    }

}
#endif

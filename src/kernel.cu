#include "hip/hip_runtime.h"
#include "kernel.cuh"

#include <cuda/std/array>
#if defined(USE_SEMAPHORE)
#include <cuda/semaphore>
#endif
#include <hiprand/hiprand_kernel.h>

#include "Circle.cuh"

__constant__ Point GLOBAL_POINTS[GLOBAL_ARRAY_SIZE];
__constant__ size_t GLOBAL_POINTS_SIZE;
__constant__ float ERROR;

__device__ static cuda::std::array<Point,3> getRandomNumber();
__device__ static void count(Circle& circle);
#if defined(USE_SEMAPHORE)
__device__  cuda::binary_semaphore<cuda::thread_scope_device> binarySemaphore{1};
__device__ static void max(Circle* bestCircle, const Circle& circle);
#endif

__global__ void ransac_kernel(Circle* bestCircle) {
    const cuda::std::array<Point,3>  randomPoints = getRandomNumber();
    Circle circle = Circle::CircleFromThreePoints(randomPoints);
    count(circle);
#if defined(USE_SEMAPHORE)
    max(bestCircle, circle);
#endif
}

__device__ static cuda::std::array<Point,3> getRandomNumber() {
    hiprandState state;
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(clock64(), id, 0, &state);
    int index1 = -1, index2 = -1, index3 = -1;

    index1 = hiprand(&state) % GLOBAL_POINTS_SIZE;

    do{
        index2 = hiprand(&state) % GLOBAL_POINTS_SIZE;
    } while (index1 == index2);

    do {
        index3 = hiprand(&state) % GLOBAL_POINTS_SIZE;
    } while (index3 == index1 || index3 == index2);

    return {
            GLOBAL_POINTS[index1],
            GLOBAL_POINTS[index2],
            GLOBAL_POINTS[index3]
    };
}

__device__ static void count(Circle& circle){
    int db = 0;
    for(size_t i = 0; i < GLOBAL_POINTS_SIZE; ++i){
        if (circle.is_point_supported(GLOBAL_POINTS[i], ERROR)) ++db;
    }
    circle.setSupportedPoints(db);
}

#if defined(USE_SEMAPHORE)
__device__ static void max(Circle* bestCircle, const Circle& circle){
    binarySemaphore.acquire();

    if (bestCircle->getSupportedPoints() < circle.getSupportedPoints()){
        *bestCircle = circle;
    }

    binarySemaphore.release();
}
#endif

#include "hip/hip_runtime.h"
#include "Ransac.cuh"

#include <fstream>
#include <iterator>
#include <fmt/ranges.h>
#include <limits>
#include <opencv2/core/mat.hpp>
#include <opencv2/core/types.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgproc.hpp>

#include "hipError_t.cuh"
#include "kernel.cuh"

constexpr static size_t point_size = sizeof(Point);

Ransac::Ransac(int iteration, float error) : iteration(iteration), error(error) {}

void Ransac::read_file(const char *filename, std::vector<Point> &dataPoints) {
    std::ifstream inputstream(filename);
    dataPoints.assign(std::istream_iterator<Point>(inputstream), std::istream_iterator<Point>());
}

void Ransac::write_to_image(const char *filename, const std::vector<Point> &points) {
    cv::Mat mat = cv::Mat::zeros(1920, 2560, CV_8UC3);

    const cv::Point2f  point {bestCircle.getCenter().getX(),bestCircle.getCenter().getY()};
    cv::circle(mat, point, bestCircle.getRadius(), cv::Scalar( 0, 0, 255 ));

    for (const Point &p: points) {
        cv::Vec3b pixel;
        pixel.val[0] = 255;
        pixel.val[1] = 0,
                pixel.val[1] = 0;
        mat.at<cv::Vec3b>(p.getY(),p.getX())=pixel;
    }

    cv::imwrite(filename,mat);
}


std::pair<int, int> Ransac::getDeviceInfo() {
    constexpr int int_min = std::numeric_limits<int>::min();
    std::pair<int, int> deviceInfo{int_min, int_min};
    int deviceCount = 0;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        fmt::print(stderr, "There are no available device(s) that support CUDA\n");
        exit(EXIT_FAILURE);
    }
#ifdef DEBUG
    else {
        fmt::print("Detected {} CUDA Capable device(s)\n", deviceCount);
    }
#endif
    for (int dev = 0; dev < deviceCount; ++dev) {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        if (deviceProp.maxThreadsPerBlock > deviceInfo.second) {
            deviceInfo.first = dev;
            deviceInfo.second = deviceProp.maxThreadsPerBlock;
        }
#ifdef DEBUG
        fmt::print("Device {}: {} \n", dev, deviceProp.name);
        fmt::print("Compute capability: {}.{}\n", deviceProp.major, deviceProp.minor);
        fmt::print("MaxThreadsPerBlock: {} \n", deviceProp.maxThreadsPerBlock);
        fmt::print("MaxThreadDim ({},{},{})  \n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
                   deviceProp.maxThreadsDim[2]);
        fmt::print("MaxGridSize ({},{},{})  \n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
                   deviceProp.maxGridSize[2]);
#endif
    }

    CUDA_CHECK(hipDeviceReset());
    return deviceInfo;
}

void Ransac::run(const char *filename) {
    std::vector<Point> points;
    read_file(filename, points);
#ifdef DEBUG
    fmt::print("Points = {}\n", points);
#endif
    const auto &[dev, maxThreadsPerBlock] = getDeviceInfo();
    hipSetDevice(dev);

    size_t byte = points.size() > GLOBAL_ARRAY_SIZE ? GLOBAL_ARRAY_SIZE * point_size : points.size() * point_size;
    size_t numberofelements = points.size();
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(GLOBAL_POINTS), points.data(), byte));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(GLOBAL_POINTS_SIZE), &numberofelements, sizeof(numberofelements)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(ERROR), &error, sizeof(error)));

#if defined(USE_SEMAPHORE)
    Circle* circle;
    hipMallocManaged(&circle, sizeof(Circle));
    *circle = Circle{};
#else
    Circle* circle;
    CUDA_CHECK(hipMalloc(&circle, iteration * maxThreadsPerBlock * sizeof(Circle)));
#endif

    ransac_kernel<<<iteration, maxThreadsPerBlock>>>(circle);
    CUDA_CHECK(hipDeviceSynchronize());

#if defined(USE_SEMAPHORE)
    bestCircle = *circle;
#else
    int dbThreads = maxThreadsPerBlock;
    int dbBlock = iteration;

    while (dbBlock > 1){
        max_search<<<dbBlock, dbThreads, dbThreads * sizeof(Circle)>>>(circle);
        CUDA_CHECK(hipDeviceSynchronize());
        dbBlock /= dbThreads;
    }
    max_search<<<1, dbThreads, dbThreads * sizeof(Circle)>>>(circle);
    CUDA_CHECK(hipMemcpy(&bestCircle, circle, sizeof(Circle), hipMemcpyDeviceToHost));
#endif

    hipFree(circle);

    fmt::print("The best circle: {}", bestCircle);
    write_to_image("test.png", points);
}
